#include "hip/hip_runtime.h"
// nvcc sigmoid.cu && ./a.out
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void multiply(int a, int *X, int *output, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) { output[id] = a * X[id]; }
    // 1 read + 1 write
}


__global__ void add(int *vector1, int *vector2, int size, int *output) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) { output[id] = vector1[id] + vector[id]; }
    // 2 read + 1 write
}


void unfused_op(int a, int *X, int *B, int size, int *output) {
    int *temp;
    hipMalloc(&temp, size * sizeof(int));
    multiply<<<1, size>>>(a, X, temp, size);
    add<<<1, size>>>(temp, B, size, output);
    hipFree(temp);

    // temp buffer is allocated (extra)
    // 3 read + 2 write => 5
}


__global__ void fused_op(int a, int *X, int *B, int size, int *output) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) { output[id] = a * X[id] + B[id]; }
    // 2 reads + 1 write => 3
}


__global__ void print(int *input, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", input[i]);
    }
    printf("\n");
}


void main() {
    int size;
    cin >> size;

    int *X = (int *) malloc(size * sizeof(int));
    int *B = (int *) malloc(size * sizeof(int));

    // generate dummy inputs
    for (int i = 0; i <  size; i++) {
        X[i] = i;
        B[i] = i * 2;
    }
    int a = 32;

    int *d_X, *d_B, *d_output;
    hipMalloc(&d_X, size * sizeof(int));
    hipMalloc(&d_B, size * sizeof(int));
    hipMalloc(&d_output, size * sizeof(int));

    hipMemcpy(d_X, X, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    unfused_op(a, d_X, d_B, size, d_output);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by unfused op is: %.6f ms\n", milliseconds);

    print<<<1, 1>>>(d_output, size);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start, 0);

    fused_op<<<1, size>>>(a, d_X, d_B, size, d_output);
    hipDeviceSynchronize();

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by fused op is: %.6f ms\n", milliseconds);

    print<<<1, 1>>>(d_output, size); 
}
