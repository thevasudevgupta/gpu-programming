#include "hip/hip_runtime.h"
// nvcc kernel_fusion.cu && ./a.out
#include <iostream>
#include <hip/hip_runtime.h>


__global__ kernel1(int *b, int *c) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) { c[id] = b[id]; }
}

__global__ kernel2(int *c, int *d) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id > 0 && id < size) { d[id] = c[id - 1]; }
}

__global__ fused_kernel(int *b, int *c, int *d) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        c[id] = b[id];

        if (id > 0) { d[id] = c[id - 1]; }
    }
}

__global__ fused_kernel(int *b, int *c, int *d) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        c[id] = b[id];

        __syncthreads();
        if (id > 0) { d[id] = c[id - 1]; }
    }
}


__global__ void multiply(int a, int *X, int *output, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) { output[id] = a * X[id]; }
    // 1 read + 1 write
}


__global__ void add(int *vector1, int *vector2, int size, int *output) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) { output[id] = vector1[id] + vector2[id]; }
    // 2 read + 1 write
}


void unfused_op(int a, int *X, int *B, int size, int *output) {
    int *temp;
    hipMalloc(&temp, size * sizeof(int));
    int num_blocks = ceil(float(size) / 1024);
    multiply<<<num_blocks, 1024>>>(a, X, temp, size);
    add<<<num_blocks, 1024>>>(temp, B, size, output);
    hipFree(temp);

    // temp buffer is allocated (extra)
    // 3 read + 2 write => 5
}


__global__ void fused_op(int a, int *X, int *B, int size, int *output) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) { output[id] = a * X[id] + B[id]; }
    // 2 reads + 1 write => 3
}


__global__ void print(int *input, int size) {
    for (int i = 0; i < size; i++) {
        printf("%d ", input[i]);
    }
    printf("\n");
}


int main() {
    int size = 100000000;

    int *X = (int *) malloc(size * sizeof(int));
    int *B = (int *) malloc(size * sizeof(int));

    // generate dummy inputs
    for (int i = 0; i <  size; i++) {
        X[i] = i;
        B[i] = i * 2;
    }
    int a = 32;

    int *d_X, *d_B, *d_output;
    hipMalloc(&d_X, size * sizeof(int));
    hipMalloc(&d_B, size * sizeof(int));
    hipMalloc(&d_output, size * sizeof(int));

    hipMemcpy(d_X, X, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop; hipEventCreate(&start);
    hipEventCreate(&stop); float milliseconds = 0;
    hipEventRecord(start,0);

    unfused_op(a, d_X, d_B, size, d_output); hipDeviceSynchronize();

    hipEventRecord(stop, 0); hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by unfused op is: %.6f ms\n", milliseconds);

    // print<<<1, 1>>>(d_output, size); hipDeviceSynchronize();

    hipEvent_t start2, stop2; hipEventCreate(&start2);
    hipEventCreate(&stop2); float milliseconds2 = 0;
    hipEventRecord(start2, 0);

    int num_blocks = ceil(float(size) / 1024);
    fused_op<<<num_blocks, 1024>>>(a, d_X, d_B, size, d_output); hipDeviceSynchronize();

    hipEventRecord(stop2,0); hipEventSynchronize(stop2);
    hipEventElapsedTime(&milliseconds2, start2, stop2);
    printf("Time taken by fused op is: %.6f ms\n", milliseconds2);

    // print<<<1, 1>>>(d_output, size); hipDeviceSynchronize();
}
