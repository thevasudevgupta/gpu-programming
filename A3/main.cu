// nvcc main.cu && ./a.out sample.txt output.txt
// nvcc main.cu && ./a.out Evaluation_script/testcases/input/input1.txt output.txt && python3 verify.py Evaluation_script/testcases/output/output1.txt output.txt

#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;


__device__ volatile int t = 0;


__global__ void initialize(volatile int *array, int size, int value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) { array[idx] = value; }
}


__global__ void simulate(volatile int *task_schedule_status, int *priority, int *executionTime,  volatile int *priority_to_core_map, volatile int *core_free_status, volatile  int *core_busy_time, int *result, int n, int m) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > n) { return; }

    // current task can't execute untill the previous task is already scheduled
    while (idx > 0 && task_schedule_status[idx - 1] == 0) ;

    int p = priority[idx];
    int core_idx = priority_to_core_map[p];
    // task is not allocated any core yet! let's allocate core then!
    if (core_idx == -1) {

        // find available core with min core idx
        int tmp_core_idx = 0;
        while (tmp_core_idx < m && core_free_status[tmp_core_idx] == 1) { tmp_core_idx += 1; }
        if (tmp_core_idx > m) { core_idx = -1; }
        else { core_idx = tmp_core_idx; }

        priority_to_core_map[p] = core_idx;
    }

    if (core_free_status[core_idx] == 1) {
        t = core_busy_time[core_idx];

        // free the cores whenever needed
        for (int i = 0; i < m; i++) {
            if (t >= core_busy_time[i]) {
                core_free_status[i] = 0;
            }
        }
    }

    result[idx] = t + executionTime[idx];
    // we want all the tasks to wait until that core is free
    // printf("t=%d | task-%d is scheduled on core-%d untill %d units time\n", t, idx, core_idx, result[idx]);

    core_busy_time[core_idx] = result[idx];
    core_free_status[core_idx] = 1;

    task_schedule_status[idx] = 1; // unlock next thread
}


//Complete the following function
void operations ( int m, int n, int *executionTime, int *priority, int *result )  {
    // m -> no of cores
    // n -> no of tasks
    // exectutionTime -> {task: execution time} ; shape: n
    // priority -> {task: priority} ; shape: n
    // result -> {task: end time} ; shape: n

    // allocating memory on GPU
    int *d_executionTime, *d_priority, *d_result;
    hipMalloc(&d_executionTime, n * sizeof(int));
    hipMalloc(&d_priority, n * sizeof(int));
    hipMalloc(&d_result, n * sizeof(int));

    // copy arrays from CPU to GPU
    hipMemcpy(d_executionTime, executionTime, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_priority, priority, n * sizeof(int), hipMemcpyHostToDevice);

    // ###################################################################
    int num_blocks;

    volatile int *d_task_schedule_status;
    hipMalloc(&d_task_schedule_status, n * sizeof(int));
    num_blocks = ceil(float(n) / 1024);
    initialize<<<num_blocks, 1024>>>(d_task_schedule_status, n, 0);
    hipDeviceSynchronize();
    // 0 -> task is not scheduled yet
    // 1 -> task has been scheduled

    volatile int *d_priority_to_core_map;
    hipMalloc(&d_priority_to_core_map, m * sizeof(int));
    num_blocks = ceil(float(m) / 1024);
    initialize<<<num_blocks, 1024>>>(d_priority_to_core_map, m, -1);
    hipDeviceSynchronize();

    volatile int *d_core_free_status;
    hipMalloc(&d_core_free_status, m * sizeof(int));
    num_blocks = ceil(float(m) / 1024);
    initialize<<<num_blocks, 1024>>>(d_core_free_status, m, 0);
    hipDeviceSynchronize();

    volatile int *d_core_busy_time;
    hipMalloc(&d_core_busy_time, m * sizeof(int));
    num_blocks = ceil(float(m) / 1024);
    initialize<<<num_blocks, 1024>>>(d_core_busy_time, m, 0);
    hipDeviceSynchronize();
    // ###################################################################

    // TODO: think can we have threads within same warp?
    // num_blocks = ceil(float(n) / 1024);
    // simulate<<<num_blocks, 1024>>>(d_task_schedule_status, d_priority, d_executionTime, d_priority_to_core_map, d_core_free_status, d_core_busy_time, d_result, n, m);
    simulate<<<n, 1>>>(d_task_schedule_status, d_priority, d_executionTime, d_priority_to_core_map, d_core_free_status, d_core_busy_time, d_result, n, m);
    hipDeviceSynchronize();

    // copy results back to host
    hipMemcpy(result, d_result, n * sizeof(int), hipMemcpyDeviceToHost);

    // free up GPU memory
    hipFree(d_executionTime);
    hipFree(d_priority);
    hipFree(d_result);
}


int main(int argc,char **argv)
{
    int m,n;
    //Input file pointer declaration
    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");
    
    //Checking if file ptr is NULL
    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &m );      //scaning for number of cores
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of tasks
   
   //Taking execution time and priorities as input	
    int *executionTime = (int *) malloc ( n * sizeof (int) );
    int *priority = (int *) malloc ( n * sizeof (int) );
    for ( int i=0; i< n; i++ )  {
            fscanf( inputfilepointer, "%d", &executionTime[i] );
    }

    for ( int i=0; i< n; i++ )  {
            fscanf( inputfilepointer, "%d", &priority[i] );
    }

    //Allocate memory for final result output 
    int *result = (int *) malloc ( (n) * sizeof (int) );
    for ( int i=0; i<n; i++ )  {
        result[i] = 0;
    }
    
     hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    //==========================================================================================================
	

	operations ( m, n, executionTime, priority, result ); 
	
    //===========================================================================================================
    
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    
    // Output file pointer declaration
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    //Total time of each task: Final Result
    for ( int i=0; i<n; i++ )  {
        fprintf( outputfilepointer, "%d ", result[i]);
    }

    fclose( outputfilepointer );
    fclose( inputfilepointer );
    
    free(executionTime);
    free(priority);
    free(result);
    
    
    
}
