#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
// #include <iostream>

using namespace std;

// // following memory should be accessible to all threads
// priority -> allotted core
// core -> free/busy
// task -> allotted core
// result
// some tracker deciding which thread to block

// // what to do in code?


int find_useful_core(int p, int *priority_hashmap, int m, int *core_free_status) {
    int core_idx = priority_hashmap[p];

    // core is not mapped yet!
    if (core_idx == -1) {
        int idx = 0;
        while (idx < m && core_free_status[idx] == 1) { idx += 1; }

        // when no core is free
        if (idx > m) { core_idx = -1; }
        else { core_idx = idx; }
    }
    else {
        if (core_free_status[core_idx] == 1) { core_idx = -1; }
    }
    return core_idx;
}


__global__ void(int size, int *A, int *B, int *C) {
    // A = B + C
    idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) { A[idx] = B[idx] + C[idx]; }
}

__global__ void initialize(int *array, int size, int value) {
    // array[i] = value
    idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) { array[idx] = value; }
}

void initialize_state(int *d_tasks_start_time, int *d_priority_hashmap, int *d_core_free_status, int *d_task_core_mapping, int m, int n) {
    hipMalloc(&d_tasks_start_time, n * sizeof(int));
    hipMalloc(&d_priority_hashmap, m * sizeof(int));
    hipMalloc(&d_core_free_status, m * sizeof(int));
    hipMalloc(&d_task_core_mapping, n * sizeof(int));

    num_blocks = ceil(float(n) / 1024);
    initialize<<<num_blocks, 1024>>>(d_tasks_start_time, n, -1);
    initialize<<<num_blocks, 1024>>>(d_task_core_mapping, n, -1);

    num_blocks = ceil(float(m) / 1024);
    initialize<<<num_blocks, 1024>>>(d_priority_hashmap, m, -1); // {priority: core}
    initialize<<<num_blocks, 1024>>>(d_core_free_status, n, 0); // 0: free core ; 1: busy

    hipDeviceSynchronize();
}


__global__ void simulate() {
    // 
    int timeout = 0;
    for (int i = 0; i < n; i++) { timeout += executionTime[i]; }

    bool time_updated = false;
    int t = 0;
    int task_idx = 0;
    while (true) {
        if (t > timeout) { break; }
        printf("t=%d :: ", t);

        int p = priority[task_idx];

        if (time_updated) {
            for (int i = 0; i < task_idx; i++) {
                if (t == tasks_start_time[i] + executionTime[i]) {
                    // cout << "inside" << endl;
                    core_free_status[task_core_mapping[i]] = 0;
                }
            }
        }

        int core_idx = find_useful_core(p, priority_hashmap, m, core_free_status);

        // no core is available
        // task has to be blocked until free core becomes available
        if (core_idx == -1) {
            t++;
            time_updated = true;
            printf("doing nothing!!\n");
            continue;
        }

        printf("scheduling task-%d on core-%d\n", task_idx, core_idx);

        // core is busy now!
        core_free_status[core_idx] = 1;
        task_core_mapping[task_idx] = core_idx;

        // priority should be mapped to this core now!
        if (priority_hashmap[p] == -1) {
            priority_hashmap[p] = core_idx;
        }

        tasks_start_time[task_idx] = t;

        task_idx++;
        time_updated = false;

        // once all tasks are scheduled and result is computed, we can stop the script
        if (task_idx == n) { break; }
    }
    //
}


//Complete the following function
void operations ( int m, int n, int *executionTime, int *priority, int *result )  {
    // m -> no of cores
    // n -> no of tasks
    // exectutionTime -> {task: execution time} ; shape: n
    // priority -> {task: priority} ; shape: n
    // result -> {task: end time} ; shape: n

    // allocating memory on GPU
    int *d_executionTime, *d_priority, *d_result;
    hipMalloc(&d_executionTime, n * sizeof(int));
    hipMalloc(&d_priority, n * sizeof(int));
    hipMalloc(&d_result, n * sizeof(int));

    // copy arrays from CPU to GPU
    hipMemcpy(d_executionTime, executionTime, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_priority, priority, n * sizeof(int), hipMemcpyHostToDevice);

    int *d_tasks_start_time, *d_priority_hashmap, *d_core_free_status, *d_task_core_mapping;
    initialize_state(d_tasks_start_time, d_priority_hashmap, d_core_free_status, d_task_core_mapping, m, n);

    // TODO: think can we have threads within same warp?
    simulate<<<n, 1>>>();
    hipDeviceSynchronize();

    num_blocks = ceil(float(n) / 1024);
    fill_sum<<<num_blocks, 1024>>>(n, d_result, d_tasks_start_time, d_executionTime);
    hipDeviceSynchronize();

    // copy results back to host
    hipMemcpy(result, d_result, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_tasks_start_time);
    hipFree(d_priority_hashmap);
    hipFree(d_core_free_status);
    hipFree(d_task_core_mapping);

    hipFree(d_executionTime);
    hipFree(d_priority);
    hipFree(d_result);
}




int main(int argc,char **argv)
{
    int m,n;
    //Input file pointer declaration
    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");
    
    //Checking if file ptr is NULL
    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &m );      //scaning for number of cores
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of tasks
   
   //Taking execution time and priorities as input	
    int *executionTime = (int *) malloc ( n * sizeof (int) );
    int *priority = (int *) malloc ( n * sizeof (int) );
    for ( int i=0; i< n; i++ )  {
            fscanf( inputfilepointer, "%d", &executionTime[i] );
    }

    for ( int i=0; i< n; i++ )  {
            fscanf( inputfilepointer, "%d", &priority[i] );
    }

    //Allocate memory for final result output 
    int *result = (int *) malloc ( (n) * sizeof (int) );
    for ( int i=0; i<n; i++ )  {
        result[i] = 0;
    }
    
     hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    //==========================================================================================================
	

	operations ( m, n, executionTime, priority, result ); 
	
    //===========================================================================================================
    
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    
    // Output file pointer declaration
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    //Total time of each task: Final Result
    for ( int i=0; i<n; i++ )  {
        fprintf( outputfilepointer, "%d ", result[i]);
    }

    fclose( outputfilepointer );
    fclose( inputfilepointer );
    
    free(executionTime);
    free(priority);
    free(result);
    
    
    
}
