#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
using namespace std;

ofstream outfile; //the handle for printing the output

// complete the following kernel...
__global__ void per_row_column_kernel(long int *A, long int *B, long int *C,long int m, long int n){

}

// complete the following kernel...
__global__ void per_column_row_kernel(long int *A, long int *B, long int *C,long int m, long int n){

}

// complete the following kernel...
__global__ void per_element_kernel(long int *A, long int *B, long int *C,long int m, long int n){

}

/**
 * Prints any 1D array in the form of a matrix 
 * */
void printMatrix(long int *arr, long int rows, long int cols, char* filename) {

	outfile.open(filename);
	for(long int i = 0; i < rows; i++) {
		for(long int j = 0; j < cols; j++) {
			outfile<<arr[i * cols + j]<<" ";
		}
		outfile<<"\n";
	}
	outfile.close();
}

int main(int argc,char **argv){

	//variable declarations
	long int m,n;	
	cin>>m>>n;	

	//host_arrays 
	long int *h_a,*h_b,*h_c;

	//device arrays 
	long int *d_a,*d_b,*d_c;
	
	//Allocating space for the host_arrays 
	h_a = (long int *) malloc(m * n * sizeof(long int));
	h_b = (long int *) malloc(m * n * sizeof(long int));	
	h_c = (long int *) malloc(m * n * sizeof(long int));	

	//Allocating memory for the device arrays 
	hipMalloc(&d_a, m * n * sizeof(long int));
	hipMalloc(&d_b, m * n * sizeof(long int));
	hipMalloc(&d_c, m * n * sizeof(long int));

	//Read the input matrix A 
	for(long int i = 0; i < m * n; i++) {
		cin>>h_a[i];
	}

	//Read the input matrix B 
	for(long int i = 0; i < m * n; i++) {
		cin>>h_b[i];
	}

	//Transfer the input host arrays to the device 
	hipMemcpy(d_a, h_a, m * n * sizeof(long int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, m * n * sizeof(long int), hipMemcpyHostToDevice);

	long int gridDimx, gridDimy;
	//Launch the kernels 
	/**
	 * Kernel 1 - per_row_column_kernel
	 * To be launched with 1D grid, 1D block
	 * */
	gridDimx = ceil(float(m) / 1024);
	dim3 grid1(gridDimx,1,1);
	dim3 block1(1024,1,1);
	per_row_column_kernel<<<grid1,block1>>>(d_a,d_b,d_c,m,n);
	hipDeviceSynchronize();
	hipMemcpy(h_c, d_c, m * n * sizeof(long int), hipMemcpyDeviceToHost);
	printMatrix(h_c, m, n,"kernel1.txt");
	
	/**
	 * Kernel 2 - per_column_row_kernel
	 * To be launched with 1D grid, 2D block
	 * */
	gridDimx = ceil(float(n) / 1024);
	dim3 grid2(gridDimx,1,1);
	dim3 block2(32,32,1);
	per_column_row_kernel<<<grid2,block2>>>(d_a,d_b,d_c,m,n);
	hipDeviceSynchronize();
	hipMemcpy(h_c, d_c, m * n * sizeof(long int), hipMemcpyDeviceToHost);
	printMatrix(h_c, m, n,"kernel2.txt");

	/**
	 * Kernel 3 - per_element_kernel
	 * To be launched with 2D grid, 2D block
	 * */
	gridDimx = ceil(float(n) / 16);
	gridDimy = ceil(float(m) / 64);
	dim3 grid3(gridDimx,gridDimy,1);
	dim3 block3(64,16,1);
	per_element_kernel<<<grid3,block3>>>(d_a,d_b,d_c,m,n);
	hipDeviceSynchronize();
	hipMemcpy(h_c, d_c, m * n * sizeof(long int), hipMemcpyDeviceToHost);
	printMatrix(h_c, m, n,"kernel3.txt");


	return 0;
}
