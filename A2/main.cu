// nvcc main.cu && ./a.out evaluation-script/testcases/input/input1.txt output.txt

#include<iostream>
#include<sys/time.h>
#include<hip/hip_runtime.h>

using namespace std;


// write kernels here...
__global__ void transpose(int *A, int *X, int a, int b) {
	// A -> a x b
	// X -> b x a
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= a * b) { return; }
	int ii = id / b;
	int jj = id % b;

	X[jj * a + ii] = A[ii * b + jj];
}


__global__ void matmul(int *A, int *B, int *X, int a, int b, int c) {
	// A -> a x b
	// B -> b x c
	// X -> a x c
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= a * c) { return; }

	int ii = id / c;
	int jj = id % c;

	X[ii * c + jj] = 0;
	for (int kk = 0; kk < b; kk++) {
		X[ii * c + jj] += A[ii * b + kk] * B[kk * c + jj];
		// X -> fully memory coalesced
		// B -> fully memory coalesced
	}
}


__global__ void add_(int *A, int *B, int a, int b) {
	// A = A + B
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id >= a * b) { return; }
	A[id] += B[id];
	// A, B -> fully memory coalesced
}


// function to compute the output matrix
void compute(int p, int q, int r, int s, int *h_matrixA, int *h_matrixB, 
	         int *h_matrixC, int *h_matrixD, int *h_matrixX) {
	// variable declarations...
	int *d_matrixA, *d_matrixB, *d_matrixC, *d_matrixD, *d_matrixX;
	int num_blocks;

	// temporary memory for storing intermediate state of transpose
	int *temp_d_matrix;

	// memory for storing intermediate states of C @ D.T
	int *C_DT;
	hipMalloc(&C_DT, q * s * sizeof(int));

	// allocate memory...
	hipMalloc(&d_matrixA, p * q * sizeof(int));
	hipMalloc(&d_matrixB, q * p * sizeof(int));
	hipMalloc(&d_matrixC, q * r * sizeof(int));
	hipMalloc(&d_matrixD, s * r * sizeof(int));
	hipMalloc(&d_matrixX, p * s * sizeof(int));

	// memory for storing intermediate states of transpose
	hipMalloc(&temp_d_matrix, max(s * r, p * q) * sizeof(int));

	// copy the values...
	hipMemcpy(d_matrixA, h_matrixA, p * q * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixB, h_matrixB, q * p * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixC, h_matrixC, q * r * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixD, h_matrixD, s * r * sizeof(int), hipMemcpyHostToDevice);

	// call the kernels for doing required computations...

	// compute D.T and store it in temp_d_matrix
	num_blocks = ceil(float(s * r) / 1024);
	transpose<<<num_blocks, 1024>>>(d_matrixD, temp_d_matrix, s, r);
	hipDeviceSynchronize();

	// compute C@D.T and store it in C_DT
	num_blocks = ceil(float(q * s) / 1024);
	matmul<<<num_blocks, 1024>>>(d_matrixC, temp_d_matrix, C_DT, q, r, s);
	hipDeviceSynchronize();

	// B -> B.T
	num_blocks = ceil(float(p * q) / 1024);
	transpose<<<num_blocks, 1024>>>(d_matrixB, temp_d_matrix, q, p);
	hipDeviceSynchronize();

	// A = A + B.T
	num_blocks = ceil(float(p * q) / 1024);
	add_<<<num_blocks, 1024>>>(d_matrixA, temp_d_matrix, p, q);
	hipDeviceSynchronize();

	// (A + B.T) @ C @ D.T
	num_blocks = ceil(float(p * s) / 1024);
	matmul<<<num_blocks, 1024>>>(d_matrixA, C_DT, d_matrixX, p, q, s);
	hipDeviceSynchronize();

	// copy the result back...
	hipMemcpy(h_matrixX, d_matrixX, p * s * sizeof(int), hipMemcpyDeviceToHost);

	// deallocate the memory...
	hipFree(d_matrixA);
	hipFree(d_matrixB);
	hipFree(d_matrixC);
	hipFree(d_matrixD);
	hipFree(d_matrixX);
}


// function to read the input matrices from the input file
void readMatrix(FILE *inputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fscanf(inputFilePtr, "%d", &matrix[i*cols+j]);
		}
	}
}


// function to write the output matrix into the output file
void writeMatrix(FILE *outputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fprintf(outputFilePtr, "%d ", matrix[i*cols+j]);
		}
		fprintf(outputFilePtr, "\n");
	}
}


int main(int argc, char **argv) {
	// variable declarations
	int p, q, r, s;
	int *matrixA, *matrixB, *matrixC, *matrixD, *matrixX;
	struct timeval t1, t2;
	double seconds, microSeconds;

	// get file names from command line
	char *inputFileName = argv[1];
	char *outputFileName = argv[2];

	// file pointers
	FILE *inputFilePtr, *outputFilePtr;

    inputFilePtr = fopen(inputFileName, "r");
	if(inputFilePtr == NULL) {
	    printf("Failed to open the input file.!!\n"); 
		return 0;
	}

	// read input values
	fscanf(inputFilePtr, "%d %d %d %d", &p, &q, &r, &s);

	// allocate memory and read input matrices
	matrixA = (int*) malloc(p * q * sizeof(int));
	matrixB = (int*) malloc(q * p * sizeof(int));
	matrixC = (int*) malloc(q * r * sizeof(int));
	matrixD = (int*) malloc(s * r * sizeof(int));
	readMatrix(inputFilePtr, matrixA, p, q);
	readMatrix(inputFilePtr, matrixB, q, p);
	readMatrix(inputFilePtr, matrixC, q, r);
	readMatrix(inputFilePtr, matrixD, s, r);

	// allocate memory for output matrix
	matrixX = (int*) malloc(p * s * sizeof(int));

	// call compute function to get the output matrix. it is expected that 
	// the compute function will store the result in matrixX.
	gettimeofday(&t1, NULL);
	compute(p, q, r, s, matrixA, matrixB, matrixC, matrixD, matrixX);
	hipDeviceSynchronize();
	gettimeofday(&t2, NULL);

	// print the time taken by the compute function
	seconds = t2.tv_sec - t1.tv_sec;
	microSeconds = t2.tv_usec - t1.tv_usec;
	printf("Time taken (ms): %.3f\n", 1000*seconds + microSeconds/1000);

	// store the result into the output file
	outputFilePtr = fopen(outputFileName, "w");
	writeMatrix(outputFilePtr, matrixX, p, s);

	// close files
    fclose(inputFilePtr);
    fclose(outputFilePtr);

	// deallocate memory
	free(matrixA);
	free(matrixB);
	free(matrixC);
	free(matrixD);
	free(matrixX);

	return 0;
}