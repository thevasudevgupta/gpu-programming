#include "hip/hip_runtime.h"
#include<iostream>
#include<sys/time.h>
#include<hip/hip_runtime.h>
using namespace std;


// write kernels here...
__global__ void transpose_(int *A) {

}

__global__ void matmul(int *A, int *B, int *X) {

}

__global__ void add(int *A, int *B, int *X) {

}

// A -> p x q
// B -> q x p
// C -> q x r
// D -> s x r
// X -> p x s

// p x q
// p x r

// function to compute the output matrix
void compute(int p, int q, int r, int s, int *h_matrixA, int *h_matrixB, 
	         int *h_matrixC, int *h_matrixD, int *h_matrixX) {
	// variable declarations...
	int *d_matrixA, *d_matrixB, *d_matrixC, *d_matrixD, *d_matrixX;

	// allocate memory...
	hipMalloc(&d_matrixA, p * q * sizeof(int));
	hipMalloc(&d_matrixB, q * p * sizeof(int));
	hipMalloc(&d_matrixC, q * r * sizeof(int));
	hipMalloc(&d_matrixD, s * r * sizeof(int));
	hipMalloc(&d_matrixX, p * s * sizeof(int));
	hipMalloc(&tmp_d_matrix, p * max(q, r) * sizeof(int));

	// copy the values...
	hipMemcpy(d_matrixA, h_matrixA, p * q * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixB, h_matrixB, q * p * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixC, h_matrixC, q * r * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_matrixD, h_matrixD, s * r * sizeof(int), hipMemcpyHostToDevice);

	// call the kernels for doing required computations...
	transpose_<<<, >>>(d_matrixB); // B -> B.T
	transpose_<<<, >>>(d_matrixD); // D -> D.T
	add<<<, >>>(d_matrixA, d_matrixB, tmp_d_matrix); // A + B.T
	copy_<<<, >>>(tmp_d_matrix, d_matrixA);
	matmul<<<, >>>(d_matrixA, d_matrixC, tmp_d_matrix); // (A + B.T) @ C
	matmul<<<, >>>(tmp_d_matrix, d_matrixD, d_matrixX); // (A + B.T) @ C @ D.T
	hipDeviceSynchronize();

	// copy the result back...
	hipMemcpy(h_matrixX, d_matrixX, p * s * sizeof(int), hipMemcpyDeviceToHost);

	// deallocate the memory...
	hipFree(d_matrixA);
	hipFree(d_matrixB);
	hipFree(d_matrixC);
	hipFree(d_matrixD);
	hipFree(d_matrixX);
	hipFree(tmp_d_matrix);
}

// function to read the input matrices from the input file
void readMatrix(FILE *inputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fscanf(inputFilePtr, "%d", &matrix[i*cols+j]);
		}
	}
}

// function to write the output matrix into the output file
void writeMatrix(FILE *outputFilePtr, int *matrix, int rows, int cols) {
	for(int i=0; i<rows; i++) {
		for(int j=0; j<cols; j++) {
			fprintf(outputFilePtr, "%d ", matrix[i*cols+j]);
		}
		fprintf(outputFilePtr, "\n");
	}
}

int main(int argc, char **argv) {
	// variable declarations
	int p, q, r, s;
	int *matrixA, *matrixB, *matrixC, *matrixD, *matrixX;
	struct timeval t1, t2;
	double seconds, microSeconds;

	// get file names from command line
	char *inputFileName = argv[1];
	char *outputFileName = argv[2];

	// file pointers
	FILE *inputFilePtr, *outputFilePtr;

    inputFilePtr = fopen(inputFileName, "r");
	if(inputFilePtr == NULL) {
	    printf("Failed to open the input file.!!\n"); 
		return 0;
	}

	// read input values
	fscanf(inputFilePtr, "%d %d %d %d", &p, &q, &r, &s);

	// allocate memory and read input matrices
	matrixA = (int*) malloc(p * q * sizeof(int));
	matrixB = (int*) malloc(q * p * sizeof(int));
	matrixC = (int*) malloc(q * r * sizeof(int));
	matrixD = (int*) malloc(s * r * sizeof(int));
	readMatrix(inputFilePtr, matrixA, p, q);
	readMatrix(inputFilePtr, matrixB, q, p);
	readMatrix(inputFilePtr, matrixC, q, r);
	readMatrix(inputFilePtr, matrixD, s, r);

	// allocate memory for output matrix
	matrixX = (int*) malloc(p * s * sizeof(int));

	// call compute function to get the output matrix. it is expected that 
	// the compute function will store the result in matrixX.
	gettimeofday(&t1, NULL);
	compute(p, q, r, s, matrixA, matrixB, matrixC, matrixD, matrixX);
	hipDeviceSynchronize();
	gettimeofday(&t2, NULL);

	// print the time taken by the compute function
	seconds = t2.tv_sec - t1.tv_sec;
	microSeconds = t2.tv_usec - t1.tv_usec;
	printf("Time taken (ms): %.3f\n", 1000*seconds + microSeconds/1000);

	// store the result into the output file
	outputFilePtr = fopen(outputFileName, "w");
	writeMatrix(outputFilePtr, matrixX, p, s);

	// close files
    fclose(inputFilePtr);
    fclose(outputFilePtr);

	// deallocate memory
	free(matrixA);
	free(matrixB);
	free(matrixC);
	free(matrixD);
	free(matrixX);

	return 0;
}