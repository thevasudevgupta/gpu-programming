// #include <stdio.h>
// int main() {
//     printf("Hello World.\n");
//     return 0;
// }

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel() {
    printf("Hello World.\n");
}

int main() {
    dkernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}
